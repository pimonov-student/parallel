
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <cstdio>
#include <cmath>

#include <hipblas.h>

// sigmoid function
__global__ void sigmoid(float* data, size_t size)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;

    // out of array
    if (id >= size)
    {
        return;
    }

    data[id] = 1.0 / (1.0 + expf(-data[id]));
}

// fully connected layer class
class Linear
{
public:
    // constructor
    Linear(size_t input_size, size_t output_size, char* path, hipblasHandle_t handle)
    {
        FILE* fin;
        float* buffer_for_weights;

        this->input_size = input_size;
        this->output_size = output_size;
        this->handle = handle;

        hipHostMalloc(&buffer_for_weights, sizeof(float) * input_size * output_size, hipHostMallocDefault);
        hipMalloc(&this->weights, sizeof(float) * input_size * output_size);
        hipMalloc(&this->buffer_for_output, sizeof(float) * output_size);

        // read data from file in C way
        fin = std::fopen(path, "rb");
        std::fread(buffer_for_weights, sizeof(float), input_size * output_size, fin);
        std::fclose(fin);

        hipMemcpy((void*)this->weights, (void*)buffer_for_weights, sizeof(float) * input_size * output_size, hipMemcpyHostToDevice);

        hipHostFree(buffer_for_weights);
    }
    // destructor
    ~Linear()
    {
        hipFree(this->weights);
        hipFree(this->buffer_for_output);
    }

    // forward pass
    void forward(float* input, float** input_ptr)
    {
        // necessary variables for cublas function
        const float alpha = 1.0;
        const float beta = 0.0;

        hipblasSgemv(this->handle, HIPBLAS_OP_T,
                       this->input_size, this->output_size,
                       &alpha, this->weights, this->input_size,
                       input, 1,
                       &beta, this->buffer_for_output, 1);
        
        *input_ptr = this->buffer_for_output;
    }

    size_t get_output_size()
    {
        return this->output_size;
    }
private:
    size_t input_size;
    size_t output_size;
    float* weights;
    float* buffer_for_output;
    hipblasHandle_t handle;
};

class Net
{
public:
    // constructor
    Net(size_t* sizes, char** paths)
    {
        hipblasCreate(&handle);

        hipHostMalloc(&input, sizeof(float) * sizes[0], hipHostMallocDefault);
        hipMalloc(&dev_input, sizeof(float) * sizes[0]);

        this->fc1 = std::make_unique<Linear>(sizes[0], sizes[1], paths[0], handle);
        this->fc2 = std::make_unique<Linear>(sizes[1], sizes[2], paths[1], handle);
        this->fc3 = std::make_unique<Linear>(sizes[2], sizes[3], paths[2], handle);
    }
    // destructor
    ~Net()
    {
        hipblasDestroy(handle);
        hipHostFree(input);
        hipFree(dev_input);
    }

    void read_input(char* path)
    {
        // reading inputs
        FILE* fin;
        fin = std::fopen(path, "rb");
        std::fread(input, sizeof(float), 32 * 32, fin);
        std::fclose(fin);

        // copy them to device
        hipMemcpy(dev_input, input, sizeof(float) * 32 * 32, hipMemcpyHostToDevice);
    }

    // forward pass
    void forward(float* output)
    {
        float* result = nullptr;
        size_t threads = 32;
        size_t blocks;

        this->fc1->forward(dev_input, &result);
        blocks = std::ceil((float)this->fc1->get_output_size() / threads);
        sigmoid<<<blocks, threads>>>(result, this->fc1->get_output_size());

        this->fc2->forward(result, &result);
        blocks = std::ceil((float)this->fc2->get_output_size() / threads);
        sigmoid<<<blocks, threads>>>(result, this->fc2->get_output_size());

        this->fc3->forward(result, &result);
        blocks = std::ceil((float)this->fc3->get_output_size() / threads);
        sigmoid<<<blocks, threads>>>(result, this->fc3->get_output_size());

        hipMemcpy(output, result, sizeof(float), hipMemcpyDeviceToHost);
        std::cout << *output << std::endl;
    }
private:
    std::unique_ptr<Linear> fc1;
    std::unique_ptr<Linear> fc2;
    std::unique_ptr<Linear> fc3;
    float* input;
    float* dev_input;
    hipblasHandle_t handle;
};

int main()
{
    size_t sizes[4] = { 32 * 32, 16 * 16, 4 * 4, 1 };
    char* paths[3] = { "./weights/weights_fc1.bin",
                       "./weights/weights_fc2.bin",
                       "./weights/weights_fc3.bin" };

    // output result
    float output;

    // network object
    Net net(sizes, paths);
    // reading input from .bin file
    net.read_input("./weights/weights_input.bin");

    // forward pass
    net.forward(&output);

    return 0;
}